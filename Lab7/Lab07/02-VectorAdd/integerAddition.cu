/*******************************************************************************
*                                                                              *
*  integerAddition.cu -                                                        *
*                                                                              *
*   Adds two numbers on the device (GPU)                                       *
*                                                                              *
*                     Departamento de Electronica y Ciencias de la Computacion *
*                     Pontificia Universidad Javeriana - CALI                  *
*                                                                              *
*******************************************************************************/


#include <hip/hip_runtime.h>
#include  <stdio.h>

/***( Code to be executed on the device (GPU) )****************************/

/* Add two scalars */
__global__ void add ( int *augend, int *addend, int *result )
{
  *result = *augend + *addend;
}

/***( Code to be executed on the host (CPU) )******************************/

/*--( Main function )----------------------------------------------------*/
int main ( void )
{
  int augend,       /* host copies of augend, addend, result */
      addend,
      result;
  int *d_augend,
      *d_addend,
      *d_result;    /* device copies of augend, addend, result */

  int size = sizeof (int);

  /* Allocate space for device copies of augend, addend, result */
  hipMalloc ( (void **) &d_augend, size );
  hipMalloc ( (void **) &d_addend, size );
  hipMalloc ( (void **) &d_result, size );

  /* Setup input values */
  augend = 2;
  addend = 7;

  /* Copy inputs to device */
  hipMemcpy ( d_augend, &augend, size, hipMemcpyHostToDevice );
  hipMemcpy ( d_addend, &addend, size, hipMemcpyHostToDevice );

  /* Launch add () kernel on GPU */
  add <<< 1, 1 >>> ( d_augend, d_addend, d_result );

  /* Copy result back to host */
  hipMemcpy ( &result, d_result, size, hipMemcpyDeviceToHost );

  printf ( "%d + %d = %d\n", augend, addend, result );

  /* Cleanup */
  hipFree ( d_augend );
  hipFree ( d_addend );
  hipFree ( d_result );

  return ( 0 );
}
