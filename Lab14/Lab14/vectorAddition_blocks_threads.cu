/*******************************************************************************
*                                                                              *
*  vectorAddition_blocks_threads.cu -                                          *
*                                                                              *
*   Adds two vectors on the device (GPU) using several blocks and several
*   threads per block
*                                                                              *
*                     Departamento de Electronica y Ciencias de la Computacion *
*                     Pontificia Universidad Javeriana - CALI                  *
*                                                                              *
*******************************************************************************/


#include <hip/hip_runtime.h>
#include    <stdio.h>
#include    <stdlib.h>

/***( Manifest Constants )************************************************/

#define BLOCKS            8
#define THREADS_PER_BLOCK 64

#define VEC_LEN                 (BLOCKS * THREADS_PER_BLOCK)

/***( Code to be executed on the device (GPU) )****************************/

/* Add two vectors */
__global__ void add ( int *augend, int *addend, int *result )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  result [index] = augend [index] + addend [index];
}

/***( Code to be executed on the host (CPU) )******************************/

/*--( Support functions )------------------------------------------------*/

/* Initialise a vector of the given length */
void init_vect ( int *vector, int length )
{
  int i;

  for ( i = 0; i < length; i++ )
    vector[i] = i;
}

/*--( Main function )----------------------------------------------------*/
int main ( void )
{
  int *augend, *addend, *result;        /* host copies of augend, addend, result */
  int *d_augend, *d_addend, *d_result;  /* device copies of augend, addend, result */
  int size = VEC_LEN * sizeof (int);

  /* Allocate space for host copies of augend, addend, result; setup input values */
  augend = (int *) malloc ( size ); init_vect ( augend, VEC_LEN );
  addend = (int *) malloc ( size ); init_vect ( addend, VEC_LEN );
  result = (int *) malloc ( size );

  /* Allocate space for device copies of augend, addend, result */
  hipMalloc ( (void **) &d_augend, size);
  hipMalloc ( (void **) &d_addend, size);
  hipMalloc ( (void **) &d_result, size);

  /* Copy inputs to device */
  hipMemcpy ( d_augend, augend, size, hipMemcpyHostToDevice );
  hipMemcpy ( d_addend, addend, size, hipMemcpyHostToDevice );

  /* Launch add () kernel on GPU */
  add <<< VEC_LEN / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> ( d_augend, d_addend, d_result );

  /* Copy result back to host */
  hipMemcpy ( result, d_result, size, hipMemcpyDeviceToHost );

  /* Cleanup */
  hipFree ( d_augend );
  hipFree ( d_addend );
  hipFree ( d_result );

  free ( augend );
  free ( addend );
  free ( result );

  return ( 0 );
}
