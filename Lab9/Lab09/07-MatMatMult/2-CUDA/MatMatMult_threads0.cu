/*******************************************************************************
*
*  MatMatMult_threads0.cu -  A program to multiply two matrices using CUDA
*
*   Notes:            Matrices are wired and have integer elements
*                     Uses pointers to the matrices and pointer arithmetic
*                     Uses global memory; each thread computes one element
*                     of the block sub-matrix
*
*******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N         16
#define MAT_COLS  N
#define MAT_ROWS  N

/***( CUDA kernel )*******************************************************/

__global__ void MatMulKernel ( int * MA, int * MB, int * MY )
{
  int Yval = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  for ( int i = 0; i < N; i++ )
    Yval += MA[row * N + i] * MB[i * N + col];
  MY[row * N + col] = Yval;
}

/*--( Support functions )------------------------------------------------*/

/* Initialise matrices of the given size */
void init_matrices ( int *MA, int *MB, int size )
{
  int row,
      col;

  for ( row = 0; row < size; row++ )
    for ( col = 0; col < size; col++ )
    {
      MA[row * N + col] = row * MAT_COLS + col;
      MB[row * N + col] = row * MAT_COLS + col + MAT_ROWS * MAT_COLS;
    }
}

/* Print matrix of the given size */
void PrintMatrix ( int *matrix )
{
  int row,
      col;

  for ( row = 0; row < N; row++ )
  {
    for ( col = 0; col < N; col++ )
      printf ( "%7d ", matrix [row * N + col] );
    printf ( "\n" );
  }
  printf ( "\n" );
}

/*--( Main function )----------------------------------------------------*/

int main ( void )
{
  int *MA,
      *MB,
      *MY;
  int *d_MA,
      *d_MB,
      *d_MY;

  int size = N * N * sizeof(int);

  MA = (int *) malloc ( size );
  MB = (int *) malloc ( size );
  MY = (int *) malloc ( size );

  /* Initialise A, B matrices */
  init_matrices ( MA, MB, N );

  printf ( "\n16x16 A matrix is \n" );
  PrintMatrix ( MA );

  printf ( "\n16x16 B matrix is \n" );
  PrintMatrix ( MB );

  /* set MY = { {0} } */
  MY = (int *) malloc ( size );
  memset ( (void *) MY, 0, size );

  hipMalloc ( &d_MA, size );
  hipMalloc ( &d_MB, size );

  hipMemcpy ( d_MA, MA, size, hipMemcpyHostToDevice );
  hipMemcpy ( d_MB, MB, size, hipMemcpyHostToDevice );

  hipMalloc ( &d_MY, size );
  hipMemcpy ( d_MY, MY, size, hipMemcpyHostToDevice );

  dim3 BlocksPerGrid ( 1, 1 );
  dim3 ThreadsPerBlock ( N, N );
  MatMulKernel <<< BlocksPerGrid, ThreadsPerBlock >>> ( d_MA, d_MB, d_MY );

  hipMemcpy ( MY, d_MY, size, hipMemcpyDeviceToHost );

  printf ( "\n16x16 Y matrix is \n" );
  PrintMatrix ( MY );

  hipFree ( d_MA );
  hipFree ( d_MB );
  hipFree ( d_MY );

  free ( MA );
  free ( MB );
  free ( MY );
}
