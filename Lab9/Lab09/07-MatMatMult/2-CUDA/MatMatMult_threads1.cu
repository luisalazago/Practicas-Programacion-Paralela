/*******************************************************************************
*
*  MatMatMult_threads1.cu -  A program to multiply two matrices using CUDA
*
*   Notes:            Matrices are wired and have integer elements
*                     Uses pointers to the matrices and pointer arithmetic
*                     Uses global memory; each thread computes one element
*                     of the block sub-matrix
*                     Prevent in excess threads to perform out-of-bounds
*                     operations
*
*******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N           16
#define MAT_COLS    N
#define MAT_ROWS    N

#define MAX_THREADS 512

/***( CUDA kernel )*******************************************************/

__global__ void MatMulKernel ( int * MA, int * MB, int * MY )
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  int Yval = 0;

  /* prevent in excess threads to perform out-of-bounds operations */
  if ( row < N && col < N )
  {
    for ( int i = 0; i < N; i++ )
      Yval += MA[row * N + i] * MB[i * N + col];
  }
  MY[row * N + col] = Yval;
}

/*--( Support functions )------------------------------------------------*/

/* Initialise matrices of the given size */
void init_matrices ( int *MA, int *MB, int size )
{
  int row,
      col;

  for ( row = 0; row < size; row++ )
    for ( col = 0; col < size; col++ )
    {
      MA[row * N + col] = row * MAT_COLS + col;
      MB[row * N + col] = row * MAT_COLS + col + MAT_ROWS * MAT_COLS;
    }
}

/* Print matrix of the given size */
void PrintMatrix ( int *matrix )
{
  int row,
      col;

  for ( row = 0; row < N; row++ )
  {
    for ( col = 0; col < N; col++ )
      printf ( "%7d ", matrix [row * N + col] );
    printf ( "\n" );
  }
  printf ( "\n" );
}

/*--( Main function )----------------------------------------------------*/

int main ( void )
{
  int *MA,
      *MB,
      *MY;
  int *d_MA,
      *d_MB,
      *d_MY;

  int size = N * N * sizeof(int);

  MA = (int *) malloc ( size );
  MB = (int *) malloc ( size );
  MY = (int *) malloc ( size );

  /* Initialise A, B matrices */
  init_matrices ( MA, MB, N );

  printf ( "\n16x16 A matrix is \n" );
  PrintMatrix ( MA );

  printf ( "\n16x16 B matrix is \n" );
  PrintMatrix ( MB );

  /* set MY = { {0} } */
  MY = (int *) malloc ( size );
  memset ( (void *) MY, 0, size );

  hipMalloc ( &d_MA, size );
  hipMalloc ( &d_MB, size );

  hipMemcpy ( d_MA, MA, size, hipMemcpyHostToDevice );
  hipMemcpy ( d_MB, MB, size, hipMemcpyHostToDevice );

  hipMalloc ( &d_MY, size );
  hipMemcpy ( d_MY, MY, size, hipMemcpyHostToDevice );

  dim3 BlocksPerGrid ( 1, 1 );
  dim3 ThreadsPerBlock ( N, N );
  if ( N * N > MAX_THREADS )
  {
    ThreadsPerBlock.x = MAX_THREADS;
    ThreadsPerBlock.y = MAX_THREADS;
    BlocksPerGrid.x = ceil ( (double) N / (double) ThreadsPerBlock.x );
    BlocksPerGrid.y = ceil ( (double) N / (double) ThreadsPerBlock.y );
  }

  MatMulKernel <<< BlocksPerGrid, ThreadsPerBlock >>> ( d_MA, d_MB, d_MY );

  hipMemcpy ( MY, d_MY, size, hipMemcpyDeviceToHost );

  printf ( "\n16x16 Y matrix is \n" );
  PrintMatrix ( MY );

  hipFree ( d_MA );
  hipFree ( d_MB );
  hipFree ( d_MY );

  free ( MA );
  free ( MB );
  free ( MY );
}
