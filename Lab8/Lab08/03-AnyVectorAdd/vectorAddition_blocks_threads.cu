/*******************************************************************************
*                                                                              *
*  vectorAddition_blocks_threads.cu -                                          *
*                                                                              *
*   Adds any two vectors on the device (GPU) using several blocks and several  *
*   threads per block                                                          *
*                                                                              *
*                     Departamento de Electronica y Ciencias de la Computacion *
*                     Pontificia Universidad Javeriana - CALI                  *
*                                                                              *
*******************************************************************************/


#include <hip/hip_runtime.h>
#include    <stdio.h>
#include    <stdlib.h>

/***( Manifest Constants )************************************************/

#define N                 16 * 1024 * 1024
#define BLOCKS            128
#define THREADS_PER_BLOCK 128

/***( Code to be executed on the device (GPU) )****************************/

/* Add two vectors */
__global__ void add ( int *augend, int *addend, int *result )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  while ( index < N )
  {
    result [index] = augend [index] + addend [index];
    /* shift by the total number of threads in the grid */
    index += BLOCKS * THREADS_PER_BLOCK;
  }
}

/***( Code to be executed on the host (CPU) )******************************/

/*--( Support functions )------------------------------------------------*/

/* Initialise a vector of the given length */
void init_vect ( int *vector, int length )
{
  int i;

  for ( i = 0; i < length; i++ )
    vector[i] = i;
}

/*--( Main function )----------------------------------------------------*/
int main ( void )
{
  int *augend, *addend, *result;        /* host copies of augend, addend, result */
  int *d_augend, *d_addend, *d_result;  /* device copies of augend, addend, result */
  int size = N * sizeof (int);

  /* Allocate space for host copies of augend, addend, result; setup input values */
  augend = (int *) malloc ( size ); init_vect ( augend, N );
  addend = (int *) malloc ( size ); init_vect ( addend, N );
  result = (int *) malloc ( size );

  /* Allocate space for device copies of augend, addend, result */
  hipMalloc ( (void **) &d_augend, size);
  hipMalloc ( (void **) &d_addend, size);
  hipMalloc ( (void **) &d_result, size);

  /* Copy inputs to device */
  hipMemcpy ( d_augend, augend, size, hipMemcpyHostToDevice );
  hipMemcpy ( d_addend, addend, size, hipMemcpyHostToDevice );

  /* Launch add () kernel on GPU */
  add <<< BLOCKS, THREADS_PER_BLOCK >>> ( d_augend, d_addend, d_result );

  /* Copy result back to host */
  hipMemcpy ( result, d_result, size, hipMemcpyDeviceToHost );

  /* verify that the calculation is correct */
  bool success = true;

  for ( int i = 0; i < N; i++ )
  {
    if ( result[i] != ( 2 * i ) )
    {
      printf ( "Aaaargh! Result at element %d (%d) doesn't match golden ref (%d)!\n",
               i, result[i], 2 * i );
      success = false;
    }
  }
  if ( success )
    printf ( "GPU vector addition of %d elements matches golden ref\n", N );

  /* Cleanup */
  hipFree ( d_augend );
  hipFree ( d_addend );
  hipFree ( d_result );

  free ( augend );
  free ( addend );
  free ( result );

  return ( 0 );
}
