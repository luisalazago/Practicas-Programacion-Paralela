/*******************************************************************************
*                                                                              *
*  dotp_v1.cu -                                                                *
*                                                                              *
*   Calculates the dot product of two vectors, v1 and v2                       *
*   Uses shared memory in the GPU to store partial per-block dot products to   *
*   then add up them in the CPU                                                *
*                                                                              *
*                     Departamento de Electronica y Ciencias de la Computacion *
*                     Pontificia Universidad Javeriana - CALI                  *
*                                                                              *
*******************************************************************************/


#include <hip/hip_runtime.h>
#include    <stdio.h>
#include    <stdlib.h>

/***( Manifest Constants )************************************************/

#define N                 512
#define BLOCKS            8
#define THREADS_PER_BLOCK 32

__global__ void dotp ( int *v1, int *v2, int *vyb )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  /* declare shm_bdp in the shared memory */
  __shared__ int shm_bdp[THREADS_PER_BLOCK];

  int   tdp = 0;
  while ( index < N )
  {
    tdp += v1[index] * v2[index];
    index += BLOCKS * THREADS_PER_BLOCK;  // increase by the total number of thread in a grid
  }

  /* set the shm_bdp values */
  shm_bdp[threadIdx.x] = tdp;

  /* synchronise threads in this block */
  __syncthreads();

  /* for reductions, THREADS_PER_BLOCK must be a power of 2 because of the following code */
  int i = THREADS_PER_BLOCK / 2;
  while ( i != 0 )
  {
    if ( threadIdx.x < i )
        shm_bdp[threadIdx.x] += shm_bdp[threadIdx.x + i];
    __syncthreads();
    i /= 2;
  }

  /* thread 0 writes back to the global memory */
  if ( threadIdx.x == 0 )
    vyb[blockIdx.x] = shm_bdp[0];
}

/***( Code to be executed on the host (CPU) )******************************/

/*--( Support functions )------------------------------------------------*/

/* Initialise vectors of the given length */
void init_vectors ( int *v1, int *v2, int length )
{
  int i;

  for ( i = 0; i < length; i++ )
  {
    v1[i] = i;
    v2[i] = 2 * i;
  }
}

/*--( Main function )----------------------------------------------------*/
int main ( void )
{
  int *v1, *v2, *vyb;        /* host copies of v1, v2, vyb */
  int *d_v1, *d_v2, *d_vyb;  /* device copies of v1, v2, vyb */
  int size = N * sizeof (int);

  /* Allocate space for host copies of v1, v2, vyb; setup input values */
  v1 = (int *) malloc ( size );
  v2 = (int *) malloc ( size );
  init_vectors ( v1, v2, N );
  vyb = (int *) malloc ( BLOCKS * sizeof(int) );

  /* Allocate space for device copies of v1, v2, vyb */
  hipMalloc ( (void **) &d_v1, size );
  hipMalloc ( (void **) &d_v2, size );
  hipMalloc ( (void **) &d_vyb, BLOCKS * sizeof(int) );

  /* Copy inputs to device */
  hipMemcpy ( d_v1, v1, size, hipMemcpyHostToDevice );
  hipMemcpy ( d_v2, v2, size, hipMemcpyHostToDevice );

  /* Launch dotp () kernel on GPU */
  dotp <<< BLOCKS, THREADS_PER_BLOCK >>> ( d_v1, d_v2, d_vyb );

  /* Copy vyb back to host */
  hipMemcpy ( vyb, d_vyb, BLOCKS * sizeof(int), hipMemcpyDeviceToHost );

  /* add the elements in the vyb vector */
  int result = 0;
  for ( int i = 0; i < BLOCKS; i++ )
    result += vyb[i];

  /* verify that the calculation is correct */
  bool success = true;
  #define sum_squares(x)  (int) ( (x) * ( (x) + 1 ) * ( 2 * (x) + 1 ) / 6 )
  if ( result != 2 * sum_squares ( N - 1 ) )
    success = false;
  if ( success )
    printf ( "GPU-CPU dot product (%d) matches golden ref (%d)\n", result, 2 * sum_squares ( N - 1 ) );

  /* Cleanup */
  hipFree ( d_v1 );
  hipFree ( d_v2 );
  hipFree ( d_vyb );

  free ( v1 );
  free ( v2 );
  free ( vyb );

  return ( 0 );
}
